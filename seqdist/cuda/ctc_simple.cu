#include "hip/hip_runtime.h"
__device__ __forceinline__ FLOAT max2(FLOAT a, FLOAT a1) {
    return a > a1 ? a : a1; 
}

__device__ __forceinline__ FLOAT logsumexp2(FLOAT a, FLOAT a1) {
    return a > a1 ? a + log1p(exp(a1 - a)) : a1 + log1p(exp(a - a1));
}

__device__ __forceinline__ FLOAT add(FLOAT a, FLOAT b) {return a + b;}
__device__ __forceinline__ FLOAT mul(FLOAT a, FLOAT b) {return a * b;}

extern "C" __global__ void fwd_bwd_logspace(
    FLOAT* __restrict__ alpha, FLOAT* __restrict__ beta_T,
    FLOAT* __restrict__ beta_stay, FLOAT* __restrict__ beta_move, 
    const FLOAT* __restrict__ stay_scores, const FLOAT* __restrict__ move_scores,
    int T, int N, int L
) {
    int bx = blockIdx.x, tx = threadIdx.x;
    if (tx >= L) return;
    extern __shared__ FLOAT smem[];
    if (blockIdx.y == 0) {
        FLOAT a = ZERO, a1 = ZERO;
        a = alpha[bx * L + tx];
        for (int t = 0; t < T; t++) {
            FLOAT *buf = smem + (t % 2) * blockDim.x;
            buf[tx] = a; __syncthreads(); 
            if (tx > 0) {a1 = MUL(move_scores[(t * N + bx) * (L - 1) + tx - 1], buf[tx - 1]);}
            a = SUM(MUL(stay_scores[(t * N + bx) * L + tx], a), a1);
            alpha[((t + 1) * N + bx) * L + tx] = a;
        }
    }
    else {
        FLOAT b = ZERO, b1 = ZERO;
        b = beta_T[bx * L + tx];
        for (int t = T; t > 0; t--) {
            FLOAT *buf = smem + (t % 2) * blockDim.x;
            buf[tx] = b; __syncthreads();
            if (tx < L - 1) {
                b1 = MUL(buf[tx + 1], move_scores[(((t - 1) * N + bx) * (L - 1)) + tx]);
                beta_move[((t - 1) * N + bx) * L + tx] = b1;
            }
            b = MUL(b, stay_scores[(((t - 1) * N + bx) * L) + tx]);
            beta_stay[((t - 1) * N + bx) * L + tx] = b;
            b = SUM(b, b1);
        }
    }
  }

extern "C" __global__ void fwd_bwd_logspace_loop(
    FLOAT* __restrict__ alpha, FLOAT* __restrict__ beta,
    FLOAT* __restrict__ beta_stay, FLOAT* __restrict__ beta_move, 
    const FLOAT* __restrict__ stay_scores, const FLOAT* __restrict__ move_scores,
    int T, int N, int L
) {
    int bx = blockIdx.x, tx = threadIdx.x;
    if (blockIdx.y == 0) {
        FLOAT a;
        for (int t = 0; t < T; t++) {
            for (int j = tx; j < L; j += blockDim.x) {
                a = (j > 0) ? MUL(move_scores[(t * N + bx) * (L - 1) + j - 1], alpha[(t * N + bx) * L + j - 1]) : ZERO;
                alpha[((t + 1) * N + bx) * L + j] = SUM(MUL(stay_scores[(t * N + bx) * L + j], alpha[(t * N + bx) * L + j]), a);
            }
            __syncthreads();
        }
    }
    else {
        FLOAT b, b1;
        for (int t = T; t > 0; t--) {
            for (int j = L - blockDim.x + tx; j >= 0; j -= blockDim.x) {
                b1 = ZERO;
                if (j < L - 1) {
                    b1 = MUL(beta[(t * N + bx) * L + j + 1], move_scores[(((t - 1) * N + bx) * (L - 1)) + j]);
                    beta_move[((t - 1) * N + bx) * L + j] = b1;
                }
                b = MUL(beta[(t * N + bx) * L + j], stay_scores[(((t - 1) * N + bx) * L) + j]);
                beta_stay[((t - 1) * N + bx) * L + j] = b;
                beta[((t - 1) * N + bx) * L + j] = SUM(b, b1);
            }
            __syncthreads();
        }
    }
  }